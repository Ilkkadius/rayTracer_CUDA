#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <chrono>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "initializers.hpp"
#include "vector3D.hpp"
#include "tracerf.hpp"
#include "rayf.hpp"
#include "backgroundsf.hpp"
#include "auxiliaryf.hpp"
#include "targetf.hpp"
#include "geometria.hpp"

// PPC
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

//###############################################################

__global__ void render(sf::Uint8 *pixels, 
        int width, int height, 
        int depth, int samples,
        targetList** list,
        BackgroundColor** background, 
        WindowVectors* window, 
        hiprandState* randState) {
            
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i >= width || j >= height) return;

    int idx = width * j + i;
    hiprandState rand = randState[idx];

    

    Vector3D color = 255 * TracePixelRnd(window, i, j, list, depth, samples, *background, rand);

    

    pixels[4*idx] = color.x;
    pixels[4*idx + 1] = color.y;
    pixels[4*idx + 2] = color.z;
    pixels[4*idx + 3] = 255;
}

__global__ void initializeBG(BackgroundColor** background) {
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        *background = createDayTime();
    }
}

__global__ void initializeTargets(Target** targets, targetList** list, Shape** shapes, int capacity) {
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        createTargets(targets, list, shapes, capacity);
    }
}

__global__ void initializeRand(hiprandState* randState, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i >= width || j >= height) return;
    int idx = i + width * j;
    hiprand_init(1889, idx, 0, &randState[idx]);
}


__global__ void releaseBG(BackgroundColor** background) {
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        delete *background;
    }
}

__global__ void releaseTargets(Target** targets, targetList** list, Shape** shapes) {
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        targetList l = **list;
        for(int i = 0; i < l.size; i++) {
            delete *(targets + i);
            delete *(shapes + i);
        }
        delete *list;
    }
}

// ################################################################

// nvcc main.cu -o main -lsfml-graphics -lsfml-window -lsfml-system

int main() {
    auto start = std::chrono::high_resolution_clock::now();
    // #################################
    // # SET PROGRAM RUN PARAMETERS
    // #################################

    int width = 1920, height = 1080;
    int depth = 5, samples = 1;
    int tx = 8, ty = 8;

    WindowVectors window = initialRays(Vector3D(0,0,0), Vector3D(1,0,0),
    1.0f, Vector3D(1,1,100), height, width, 0.8);

    // #################################
    // # LOAD DATA TO DEVICE
    // #################################

    dim3 blocks(divup(width, tx), divup(height, ty));
    dim3 threads(tx, ty);

    sf::Uint8 *pixels;// = new sf::Uint8[width*height*4];
    CHECK(hipMallocManaged(&pixels, width*height*4));

    WindowVectors *cudaWindow = NULL;
    CHECK(hipMalloc(&cudaWindow, sizeof(WindowVectors)));
    CHECK(hipMemcpy(cudaWindow, &window, sizeof(WindowVectors), hipMemcpyHostToDevice));



    BackgroundColor** background_d;
    CHECK(hipMalloc(&background_d, sizeof(BackgroundColor*)));
    initializeBG<<<1,1>>>(background_d);
    CHECK(hipDeviceSynchronize());

    hiprandState *randState_d;
    CHECK(hipMalloc(&randState_d, width*height*sizeof(hiprandState)));
    initializeRand<<<blocks, threads>>>(randState_d, width, height);
    CHECK(hipDeviceSynchronize());

    targetList** list; Target** targets; Shape** shapes; int N = 4;
    CHECK(hipMalloc(&list, sizeof(targetList*)));
    CHECK(hipMalloc(&targets, N*sizeof(Target*)));
    CHECK(hipMalloc(&shapes, N*sizeof(Shape*)));
    initializeTargets<<<1,1>>>(targets, list, shapes, N);
    CHECK(hipDeviceSynchronize());


    std::cout << "Starting GPU kernel..." << std::endl;

    render<<<blocks, threads>>>(pixels, width, height, depth, samples,
                                list, background_d, cudaWindow, 
                                randState_d);
    CHECK(hipDeviceSynchronize());
    
    std::cout << "Successfully synchronized!" << std::endl;

    //######################################
    // # GENERATE IMAGE, FREE MEMORY
    //######################################

    sf::Texture texture;
    texture.create(width, height);
    texture.update(pixels);

    sf::Image image = texture.copyToImage();
    image.saveToFile("testikuvaGPU.png");


    CHECK(hipFree(cudaWindow));
    CHECK(hipFree(pixels));
    CHECK(hipFree(randState_d));

    releaseBG<<<1,1>>>(background_d);
    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(background_d));

    releaseTargets<<<1,1>>>(targets, list, shapes);
    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(targets));
    CHECK(hipFree(list));
    CHECK(hipFree(shapes));


    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();
    std::cout << "Total program runtime: " << duration << " seconds" << std::endl;

    hipDeviceReset();

    return 0;
}