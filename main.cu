#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <chrono>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "initializers.hpp"
#include "vector3D.hpp"
#include "tracerf.hpp"
#include "rayf.hpp"
#include "backgroundsf.hpp"
#include "auxiliaryf.hpp"
#include "targetf.hpp"
#include "geometria.hpp"

// PPC
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

//###############################################################

__global__ void render(sf::Uint8 *pixels, 
        int width, int height, 
        int depth, int samples,
        targetList** list,
        BackgroundColor** background, 
        WindowVectors* window, 
        hiprandState* randState) {
            
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i >= width || j >= height) return;

    int idx = width * j + i;
    hiprandState rand = randState[idx];

    if(false && i % 100 == 0 && j % 75 == 0) {
        Vector3D vec = aux::randUnitVec(&rand);
        float l = vec.length();
        if(vec.x < 0 && vec.y < 0 && vec.z < 0) {
            printf("Vector (%f, %f, %f) of length %f\n", vec.x, vec.y, vec.z, l);
        }
    }

    

    Vector3D color = 255 * TracePixelRnd(window, i, j, list, depth, samples, *background, rand);

    

    pixels[4*idx] = color.x;
    pixels[4*idx + 1] = color.y;
    pixels[4*idx + 2] = color.z;
    pixels[4*idx + 3] = 255;
}

__global__ void initializeBG(BackgroundColor** background) {
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        *background = createDayTime();
    }
}

__global__ void initializeTargets(Target** targets, targetList** list, Shape** shapes, int capacity) {
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        createTargets(targets, list, shapes, capacity);
    }
}

__global__ void initializeRand(hiprandState* randState, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i >= width || j >= height) return;
    int idx = i + width * j;
    hiprand_init(1889, idx, 0, &randState[idx]);
}


__global__ void releaseBG(BackgroundColor** background) {
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        delete *background;
    }
}

__global__ void releaseTargets(Target** targets, targetList** list, Shape** shapes) {
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        targetList l = **list;
        for(int i = 0; i < l.size; i++) {
            delete *(targets + i);
            delete *(shapes + i);
        }
        delete *list;
    }
}

// ################################################################

// nvcc main.cu -o main -lsfml-graphics -lsfml-window -lsfml-system

int main() {
    auto start = std::chrono::high_resolution_clock::now();
    // #################################
    // # SET PROGRAM RUN PARAMETERS
    // #################################

    int width = 1920, height = 1080;
    int depth = 5, samples = 100;
    int tx = 8, ty = 8;

    WindowVectors window = initialRays(Vector3D(0,0,0), Vector3D(1,0,0),
    1.0f, Vector3D(1,1,100), height, width, 0.8);

    // #################################
    // # LOAD DATA TO DEVICE
    // #################################

    dim3 blocks(divup(width, tx), divup(height, ty));
    dim3 threads(tx, ty);

    sf::Uint8 *pixels;// = new sf::Uint8[width*height*4];
    CHECK(hipMallocManaged(&pixels, width*height*4));

    WindowVectors *cudaWindow = NULL;
    CHECK(hipMalloc(&cudaWindow, sizeof(WindowVectors)));
    CHECK(hipMemcpy(cudaWindow, &window, sizeof(WindowVectors), hipMemcpyHostToDevice));
    std::cout << "Window ready" << std::endl;



    BackgroundColor** background_d;
    CHECK(hipMalloc(&background_d, sizeof(BackgroundColor*)));
    initializeBG<<<1,1>>>(background_d);
    CHECK(hipDeviceSynchronize());
    std::cout << "Background ready" << std::endl;

    hiprandState *randState_d;
    CHECK(hipMalloc(&randState_d, width*height*sizeof(hiprandState)));
    initializeRand<<<blocks, threads>>>(randState_d, width, height);
    CHECK(hipDeviceSynchronize());
    std::cout << "Random states generated" << std::endl;

    targetList** list; Target** targets; Shape** shapes; int N = 50;
    CHECK(hipMalloc(&list, sizeof(targetList*)));
    CHECK(hipMalloc(&targets, N*sizeof(Target*)));
    CHECK(hipMalloc(&shapes, N*sizeof(Shape*)));
    initializeTargets<<<1,1>>>(targets, list, shapes, N);
    CHECK(hipDeviceSynchronize());
    std::cout << "Targets generated" << std::endl;


    std::cout << "Starting GPU rendering..." << std::endl;

    render<<<blocks, threads>>>(pixels, width, height, depth, samples,
                                list, background_d, cudaWindow, 
                                randState_d);
    CHECK(hipDeviceSynchronize());
    
    std::cout << "\033[32;1mSuccessfully rendered & synchronized!\033[0m" << std::endl;

    //######################################
    // # GENERATE IMAGE, FREE MEMORY
    //######################################

    sf::Texture texture;
    texture.create(width, height);
    texture.update(pixels);

    sf::Image image = texture.copyToImage();
    image.saveToFile("testikuvaGPU.png");


    CHECK(hipFree(cudaWindow));
    CHECK(hipFree(pixels));
    CHECK(hipFree(randState_d));

    releaseBG<<<1,1>>>(background_d);
    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(background_d));

    releaseTargets<<<1,1>>>(targets, list, shapes);
    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(targets));
    CHECK(hipFree(list));
    CHECK(hipFree(shapes));


    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();
    std::cout << "Total program runtime: " << duration << " seconds" << std::endl;

    hipDeviceReset();

    return 0;
}