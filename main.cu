#include <SFML/Graphics.hpp>
#include <chrono>
#include <iostream>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "initializers.hpp"
#include "tracerf.hpp"
#include "rayf.hpp"
#include "backgroundsf.hpp"
#include "auxiliaryf.hpp"
#include "targetList.hpp"
#include "geometria.hpp"
#include "cameraf.hpp"
#include "logMethods.hpp"
#include "imageBackupf.hpp"
#include "BVHf.hpp"
#include "fileOperations.hpp"

#include "kernelSet.hpp"

#include "realtimeRenderf.hpp"

// PPC
#ifndef CHECK_FUNC
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)
#endif

#ifndef divup_FUNC
    static inline int divup(int a, int b) {
        return (a + b - 1)/b;
    }
#endif

//static inline int roundup(int a, int b) {
//    return divup(a, b) * b;
//}


// ################################################################

// nvcc main.cu -o main -lsfml-graphics -lsfml-window -lsfml-system

int main() {
    // #################################
    // # SET PROGRAM RUN PARAMETERS
    // #################################

    Camera cam;

    int width = 1920, height = 1080;
    int depth = 4, samples = 100;
    int tx = 8, ty = 8;
    bool backup = false;
    int partition = 0;
    bool realTime = true;
    bool fileRead = false;

    cam.setFOV(80.0f);

    Vector3D eye(0, 0, 0);
    Vector3D direction(1, 0, 0);
    Vector3D up = cam.direction + Vector3D(0, 0, 100);

    // #################################
    // # LOAD DATA TO DEVICE
    // #################################

    cam.width = width; cam.height = height;
    cam.depth = depth; cam.samples = samples;

    if(partition < 0 || partition > 3) {
        std::cout << "ERROR: Invalid partition value" << std::endl;
        return 1;
    }

    std::string backupBinPath(getRawDate() + "_" + getImageDimensions(width, height) 
                            + (samples > 0 ? "_N" + std::to_string(samples) : "") + "_GPU_backup.bin");
    std::string backupTextPath = "" + getRawDate() + "_" + std::to_string(width) + "x" + std::to_string(height) 
                            + (samples > 0 ? "_N" + std::to_string(samples) : "") + "_GPU_backup.txt";

    WindowVectors *cudaWindow;
    Initializer::Window(cudaWindow, cam);

    printStartInfo(cam.width, cam.height, cam.samples, cam.depth, backup);

    dim3 blocks(divup(width, tx), divup(height, ty));
    dim3 threads(tx, ty);

    sf::Uint8 *pixels;
    CHECK(hipMallocManaged(&pixels, width*height*4));

    BackgroundColor** background_d;
    Initializer::Background(&background_d);

    hiprandState *randState_d;
    Initializer::RandomStates(&randState_d, cam.width, cam.height, blocks, threads);


    targetList** list; Target** targets; Shape** shapes; int N = 2000;
    CHECK(hipMalloc(&list, sizeof(targetList*)));
    CHECK(hipMalloc(&targets, N*sizeof(Target*)));
    CHECK(hipMalloc(&shapes, N*sizeof(Shape*)));
    initializeTargets<<<1,1>>>(targets, list, shapes, N);
    CHECK(hipDeviceSynchronize());
    
    if(fileRead) {
        FileOperations::TargetsFromFile("teapot.obj", list, shapes);
        CHECK(hipDeviceSynchronize());
    }
    // Another way to add targets from file:
    Compound** compounds; size_t compoundCount;
    CHECK(hipMalloc(&compounds, sizeof(Compound*)));
    CHECK(hipDeviceSynchronize());
    //FileOperations::CompoundsFromFile("teapot.obj", compounds, compoundCount);
    //Initializer::CompoundsToTargets(compounds, 1, list, shapes);
    
    BVHTree** tree;
    Initializer::BVH(&tree, list);
    
    std::cout << "Targets generated" << std::endl;

    if(realTime) {
        realtimeRender::startCamera(cam, tree, background_d, randState_d, eye, direction, up); // TREE
        return 0;
    }

    cam.eye = eye;
    cam.direction = direction;
    cam.up = up;

    // ####################################################################

    std::cout << "\033[0;32mGPU rendering started\033[0m" << std::endl;

    auto start = std::chrono::high_resolution_clock::now();
    auto end = std::chrono::high_resolution_clock::now();
    double duration = 0;

    if(partition == 0) {

        completeRender<<<blocks, threads>>>(pixels, width, height, depth, samples, // TREE
                                tree, background_d, cudaWindow, 
                                randState_d);
        CHECK(hipDeviceSynchronize());

        if(backup) {
            Backup::fullImageToBinary(pixels, width, height);
        }

    } else if(partition == 1) {
        renderHalf<<<blocks, threads>>>(pixels, width, height, depth, samples,
                                list, background_d, cudaWindow, 
                                randState_d, true);
        CHECK(hipDeviceSynchronize());

        if(backup) {
            Backup::halfImageToBinary(backupBinPath, pixels, width, height, true);
            //Backup::halfImageToText(backupTextPath, pixels, width, height, true);
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();

        std::cout << "Half rendered, time so far: " << getDuration(duration) << std::endl;

        renderHalf<<<blocks, threads>>>(pixels, width, height, depth, samples,
                                    list, background_d, cudaWindow, 
                                    randState_d, false);
        CHECK(hipDeviceSynchronize());

        if(backup) {
            Backup::halfImageToBinary(backupBinPath, pixels, width, height, false);
            //Backup::halfImageToText(backupTextPath, pixels, width, height, false);
        }

    } else if(partition == 2) {

        renderQuarter<<<blocks, threads>>>(pixels, width, height, depth, samples,
                                list, background_d, cudaWindow, 
                                randState_d, 0);
        CHECK(hipDeviceSynchronize());

        if(backup) {
            Backup::quarterImageToBinary(backupBinPath, pixels, width, height, 0);
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();
        std::cout << "25% rendered, time so far: " << getDuration(duration) << std::endl;

        renderQuarter<<<blocks, threads>>>(pixels, width, height, depth, samples,
                                list, background_d, cudaWindow, 
                                randState_d, 1);
        CHECK(hipDeviceSynchronize());

        if(backup) {
            Backup::quarterImageToBinary(backupBinPath, pixels, width, height, 1);
        }

        auto prevDuration = duration;
        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();
        std::cout << "50% rendered, time so far: " << getDuration(duration) << " (\u0394t = " << getRawDuration(duration - prevDuration) << ")" << std::endl;

        renderQuarter<<<blocks, threads>>>(pixels, width, height, depth, samples,
                                list, background_d, cudaWindow, 
                                randState_d, 2);
        CHECK(hipDeviceSynchronize());

        if(backup) {
            Backup::quarterImageToBinary(backupBinPath, pixels, width, height, 2);
        }

        prevDuration = duration;
        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();
        std::cout << "75% rendered, time so far: " << getDuration(duration) << " (\u0394t = " << getRawDuration(duration - prevDuration) << ")" << std::endl;

        renderQuarter<<<blocks, threads>>>(pixels, width, height, depth, samples,
                                list, background_d, cudaWindow, 
                                randState_d, 3);
        CHECK(hipDeviceSynchronize());

        if(backup) {
            Backup::quarterImageToBinary(backupBinPath, pixels, width, height, 3);
        }

    }

    
    if(backup)
        Backup::fullImageToBinary(pixels, width, height);
    
    
    std::cout << "\033[32;1mSuccessfully rendered & synchronized!\033[0m" << std::endl;

    end = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();
    std::cout << "Rendertime: " << getDuration(duration) << std::endl;

    //######################################
    // # GENERATE IMAGE, FREE MEMORY
    //######################################

    sf::Texture texture;
    texture.create(width, height);
    texture.update(pixels);

    sf::Image image = texture.copyToImage();

    std::string filename = getImageFilename(width, height, samples, duration);
    if(!image.saveToFile("figures/" + filename)) {
        if(!image.saveToFile(filename)) {
            std::cout << "\033[31mImage was not saved...\033[0m" << std::endl;
        } else {
            std::cout << "\033[32;1mImage saved!\033[0m" << std::endl;
        }
    } else {
        std::cout << "\033[32;1mImage saved!\033[0m" << std::endl;
    }

    CHECK(hipFree(cudaWindow));
    CHECK(hipFree(pixels));
    CHECK(hipFree(randState_d));

    releaseBG<<<1,1>>>(background_d);
    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(background_d));

    //releaseTargets<<<1,1>>>(targets, list, shapes);
    releaseBVH<<<1,1>>>(targets, list, shapes, *tree);
    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(targets));
    CHECK(hipFree(list));
    CHECK(hipFree(shapes));
    CHECK(hipFree(tree));



    

    hipDeviceReset();

    return 0;
}